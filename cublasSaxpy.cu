#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdio.h>


__global__ void saxpy(const float *x, float *y, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        y[i] += 1.0 * x[i];
    }
}

int main(int argc, char **argv) {
    printf("[Saxpy CUBLAS] - Starting...\n");

    int N = 1024;
    float alpha = 1.0;
    float *d_x, *d_r;

    hipblasHandle_t handle;
    hipEvent_t start, stop;

    hipblasCreate(&handle);
    hipMalloc((void **)&d_x, N * sizeof(float));
    hipMalloc((void **)&d_r, N * sizeof(float));

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    printf("blocksPerGrid=%d, threadsPerBlock=%d", blocksPerGrid, threadsPerBlock);

    hipEventCreate(&start);
    hipEventCreate(&stop);


    int nIter = 30;

    hipEventRecord(start, NULL);
    for (int i = 0; i < nIter; i++) {
        hipblasSaxpy(handle, N, &alpha, d_x, 1, d_r, 1);
        saxpy<<<blocksPerGrid, threadsPerBlock>>>(d_x, d_r, N)
    }
    printf("done.\n");

    // Record the stop event
    hipEventRecord(stop, NULL);

    // Wait for the stop event to complete
    hipEventSynchronize(stop);

    float msecTotal = 0.0f;
    hipEventElapsedTime(&msecTotal, start, stop);


    float msecPerSaxpy = msecTotal / nIter;

    double gFlops = (2 * N * 1.0e-9f ) / (msecPerSaxpy / 1000.0f);
    printf("Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops\n",
           gFlops, msecPerSaxpy, 2*N);


    hipFree(d_x);
    hipFree(d_r);
    hipblasDestroy(handle);

}